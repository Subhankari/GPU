#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include<time.h>

#define SIZE 1000
#define BLKS 4
#define THREADSPBLKS 256

__global__
void heatCalcKernel(float * g_d,float * h_d, int width, int itr)
{
  int i = threadIdx.x + (blockDim.x * blockIdx.x); 
  int row = (i / width);
  int col = i % width;
  int left = i - 1;
  int right = i + 1;
  int top = ((row - 1) * width) + col;
  int bottom = ((row + 1) * width + col);

	if(((i % width) == 0) || ((i % width) == (width - 1)) || (i < width) || (i >= (width * (width - 1)))){
		h_d[i] = g_d[i];
	}else{
		h_d[i] = 0.25 * (g_d[top] + g_d[left] + g_d[bottom] + g_d[right]);
	}
	__syncthreads();
	
	g_d[i] = h_d[i];
	
	__syncthreads();
  
}

__global__
void initializeKernel(float * g_d,int width){
	int j = threadIdx.x + (blockDim.x * blockIdx.x); 

	if((j >= 10) && (j <= 30)){
        g_d[j] = 150;
    }else if((j < width) || ((j % width) == 0) || ((j % width) == (width - 1)) || (j >= (width * (width - 1)))){
        g_d[j] = 80;
    }else{
        g_d[j] = 0;
    }
}

void heatCalc()
{
	clock_t tic;
	clock_t toc;
	tic = clock();
	int width = 101; //32
	int itr = 500;
	int len = width * width;
	float inhost[len];
	float outhost[len];
	int j;
	float * g_d;
	float * h_d;
	int counter = 0;
  
  /*----------------------------------------------------------------*/
  hipError_t error;
  hipDeviceProp_t dev;
  error = hipGetDeviceProperties(&dev, 0);
     if(error != hipSuccess)
     {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
     }
     printf("\nDevice %d:\n", 0);
     printf("name: %s\n",dev.name);
  
  hipSetDevice(0);
  /*--------------------------------------------------------------*/
  hipMalloc((void**) &g_d,len*sizeof(float));
/*  
  for( j = 0; j < len; j++){
	if((j >= 10) && (j <= 30)){
        inhost[j] = 150;
    }else if((j < width) || ((j % width) == 0) || ((j % width) == (width - 1)) || (j >= (width * (width - 1)))){
        inhost[j] = 80;
    }else{
        inhost[j] = 0;
    }
 }*/
 
  dim3 dimGrid(10);
  dim3 dimBlock(1024);

  // kernel invocation
 
 
  initializeKernel<<<dimGrid,dimBlock>>>(g_d,width);
  hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
	
  hipMemcpy(inhost, g_d, (len*sizeof(float)), hipMemcpyDeviceToHost);
   err = hipGetLastError();
  if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
  
 
  for( j = 0; j < len; j++){
    outhost[j] = 0;
 }
  
  printf("---------\n");
  
  
  hipMalloc((void**)&g_d, len*sizeof(float));
  
  //intialize the matrix
 hipMemcpy(g_d,inhost,len*sizeof(float),hipMemcpyHostToDevice);
  
  hipMalloc((void**)&h_d, len*sizeof(float));

  //dim3 dimGrid(10);
  //dim3 dimBlock(1024);

  // kernel invocation
 
  for(counter = 0; counter < itr; counter++){
  heatCalcKernel<<<dimGrid,dimBlock>>>(g_d,h_d,width,itr);
   err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
	}
  hipMemcpy(outhost, h_d, (len*sizeof(float)), hipMemcpyDeviceToHost);
    err = hipGetLastError();
  if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
  
  hipFree(g_d);
  hipFree(h_d);
  for( j = 0; j < len; j++){
    inhost[j] = outhost[j];
	printf("%f\n",inhost[j]);
	}
	
	toc = clock();
	double time_taken_parallel = (double)(toc -tic)/CLOCKS_PER_SEC; // in seconds
	printf("time taken: %f\n", time_taken_parallel);
}

int main()
{
   heatCalc();
   
    return 0;
}
