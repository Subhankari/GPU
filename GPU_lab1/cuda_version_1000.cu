#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include<time.h>

#define SIZE 1000
#define BLKS 4
#define THREADSPBLKS 256

__global__
void heatCalcKernel(float * g_d,float * h_d, int width, int itr)
{
  int i = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  int row = (i / width);
  int col = i % width;
  int left = i - 1;
  int right = i + 1;
  int top = ((row - 1) * width) + col;
  int bottom = ((row + 1) * width + col);
  
	if(((i % width) == 0) || ((i % width) == (width - 1)) || (i < width) || (i >= (width * (width - 1)))){
		h_d[i] = g_d[i];
	}else{
		h_d[i] = 0.25 * (g_d[top] + g_d[left] + g_d[bottom] + g_d[right]);
	}
	__syncthreads();
	
	g_d[i] = h_d[i];
	
	__syncthreads();  
}

void heatCalc()
{
  clock_t tic;
  clock_t toc;
  tic = clock();
  int width = 1001; //32
  int itr = 50;
  int len = width * width;
  float inhost[len];
  float outhost[len];
  int j;
  float * g_d;
  float * h_d;
  int counter = 0;

  /*----------------------------------------------------------------*/
  hipError_t error;
  hipDeviceProp_t dev;
  error = hipGetDeviceProperties(&dev, 0);
     if(error != hipSuccess)
     {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
     }
     printf("\nDevice %d:\n", 0);
     printf("name: %s\n",dev.name);
  
  hipSetDevice(0);
  /*--------------------------------------------------------------*/
  
  for( j = 0; j < len; j++){
	if((j >= 10) && (j <= 30)){
        inhost[j] = 150;
    }else if((j < width) || ((j % width) == 0) || ((j % width) == (width - 1)) || (j >= (width * (width - 1)))){
        inhost[j] = 80;
    }else{
        inhost[j] = 0;
    }
 }
 
  for( j = 0; j < len; j++){
    outhost[j] = 0;
 }
 
  printf("---------\n");
  
  
  hipMalloc((void**)&g_d, len*sizeof(float));
  
  //intialize the matrix
 hipMemcpy(g_d,inhost,len*sizeof(float),hipMemcpyHostToDevice);
  
  hipMalloc((void**)&h_d, len*sizeof(float));

  dim3 dimGrid(1958);
  dim3 dimBlock(16,32);

  // kernel invocation
 
  for(counter = 0; counter < itr; counter++){
  heatCalcKernel<<<dimGrid,dimBlock>>>(g_d,h_d,width,itr);
  hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
	}

  //transfer C_d from device to host
  hipMemcpy(outhost, h_d, (len*sizeof(float)), hipMemcpyDeviceToHost);
   hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
  
  hipFree(g_d);
  hipFree(h_d);
 
 for( j = 0; j < len; j++){
    inhost[j] = outhost[j];
	}
	
	toc = clock();
	double time_taken_parallel = (double)(toc -tic)/CLOCKS_PER_SEC; // in seconds
	printf("time taken: %f\n", time_taken_parallel);

}
int main()
{
   heatCalc();
    return 0;
}
